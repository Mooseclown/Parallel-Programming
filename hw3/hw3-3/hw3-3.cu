
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <algorithm>
#include <omp.h>

#define B 64
#define THREAD_NUM 32
#define STREAM_SIZE 20

#define unlikely(x)  __builtin_expect(!!(x), 0)
#define ROUND_UP(x, align) (((int) (x) + (align - 1)) & ~(align - 1))

const int INF = ((1 << 30) - 1);

int v_num, e_num, v_orig;

inline int ceil(int a, int b) { return (a + b - 1) / b; }

static unsigned int *input(const char *filename) {
    FILE *f;
    f = fopen(filename, "rb");
    assert(f);

    fread(&v_orig, sizeof(int), 1, f);
    fread(&e_num, sizeof(int), 1, f);
    v_num = ROUND_UP(v_orig, B);
    unsigned int *G;
    hipHostMalloc(&G, v_num * v_num * sizeof(unsigned int), hipHostMallocDefault);

    /* init shortest-path distance array*/
    std::fill_n(G, v_num * v_num, INF);
    for (int i = 0; i < v_num; ++i) {
        G[i * v_num + i] = 0;
    }

    int *pairs = (int *)malloc(3 * e_num * sizeof(int));
    fread(pairs, sizeof(int), 3 * e_num, f);
    for (int i = 0; i < e_num; ++i) {
        int index = pairs[3 * i] * v_num + pairs[3 * i + 1];
        G[index] = pairs[3 * i + 2];
    }
    free(pairs);
    
    fclose(f);
    return G;
}

void output(char* outFileName, unsigned int *G) {
    FILE *f;
    f = fopen(outFileName, "wb+");
    assert(f);

    for (int i = 0; i < v_orig; ++i) {
        fwrite(G + i * v_num, sizeof(int), v_orig, f);
    }
    fclose(f);
}

__global__ void block_FW_phase1(int row_size, int round, unsigned int *dist) {
    __shared__ unsigned int s_dist[B][B];

    int block_start = round * B;

    int tid_x = threadIdx.x;
    int tid_y = threadIdx.y;

    /* pivot (global -> share) */
    #pragma unroll 2
    for (int i = 0; i < B; i += THREAD_NUM) {
        #pragma unroll 2
        for (int j = 0; j < B; j += THREAD_NUM) {
            int si = tid_y + i;
            int sj = tid_x + j;

            int global_idx = (block_start + si) * row_size + block_start + sj;

            s_dist[si][sj] = dist[global_idx];
        }
    }

    /* cal */
    #pragma unroll 64
    for (int k = 0; k < B; ++k) {
        __syncthreads();
        #pragma unroll 2
        for (int i = 0; i < B; i += THREAD_NUM) {
            #pragma unroll 2
            for (int j = 0; j < B; j += THREAD_NUM) {
                int si = tid_y + i;
                int sj = tid_x + j;
                unsigned int tmp = s_dist[si][k] + s_dist[k][sj];
                s_dist[si][sj] = min(s_dist[si][sj], tmp);
            }
        }
    }

    /* pivot (share -> global) */
    #pragma unroll 2
    for (int i = 0; i < B; i += THREAD_NUM) {
        #pragma unroll 2
        for (int j = 0; j < B; j += THREAD_NUM) {
            int si = tid_y + i;
            int sj = tid_x + j;

            int global_idx = (block_start + si) * row_size + block_start + sj;

            dist[global_idx] = s_dist[si][sj];
        }
    }
}

__global__ void block_FW_phase2_row(int row_size, int round, unsigned int *dist) {    
    __shared__ unsigned int self[B][B];
    __shared__ unsigned int pivot[B][B];

    int pivot_start = round * B;
    int block_start = blockIdx.x * B;

    int tid_x = threadIdx.x;
    int tid_y = threadIdx.y;

    /* global -> share */
    #pragma unroll 2
    for (int x = 0; x < B; x += THREAD_NUM) {
        #pragma unroll 2
        for (int y = 0; y < B; y += THREAD_NUM) {
            int sx = tid_y + x;
            int sy = tid_x + y;

            int self_idx =  (block_start + sx) * row_size + pivot_start + sy;
            int pivot_idx = (pivot_start + sx) * row_size + pivot_start + sy;

            self[sx][sy] = dist[self_idx];
            pivot[sx][sy] = dist[pivot_idx];
        }
    }

    /* cal */
    #pragma unroll 64
    for (int k = 0; k < B; ++k) {
        __syncthreads();
        #pragma unroll 2
        for (int i = 0; i < B; i += THREAD_NUM) {
            #pragma unroll 2
            for (int j = 0; j < B; j += THREAD_NUM) {
                int si = tid_y + i;
                int sj = tid_x + j;
                unsigned int tmp = self[si][k] + pivot[k][sj];
                self[si][sj] = min(self[si][sj], tmp);
            }
        }
    }
    
    /* self(i,k) (share -> global) */
    #pragma unroll 2
    for (int i = 0; i < B; i += THREAD_NUM) {
        #pragma unroll 2
        for (int k = 0; k < B; k += THREAD_NUM) {
            int si = tid_y + i;
            int sk = tid_x + k;

            int global_idx = (block_start + si) * row_size + pivot_start + sk;

            dist[global_idx] = self[si][sk];
        }
    }
}

__global__ void block_FW_phase2_column(int row_size, int round, unsigned int *dist) {
    __shared__ unsigned int self[B][B];
    __shared__ unsigned int pivot[B][B];

    int pivot_start = round * B;
    int block_start = blockIdx.x * B;

    int tid_x = threadIdx.x;
    int tid_y = threadIdx.y;

    /* global -> share */
    #pragma unroll 2
    for (int x = 0; x < B; x += THREAD_NUM) {
        #pragma unroll 2
        for (int y = 0; y < B; y += THREAD_NUM) {
            int sx = tid_y + x;
            int sy = tid_x + y;

            int self_idx = (pivot_start + sx) * row_size + block_start + sy;
            int pivot_idx = (pivot_start + sx) * row_size + pivot_start + sy;

            self[sx][sy] = dist[self_idx];
            pivot[sx][sy] = dist[pivot_idx];
        }
    }

    /* cal */
    #pragma unroll 64
    for (int k = 0; k < B; ++k) {
        __syncthreads();
        #pragma unroll 2
        for (int i = 0; i < B; i += THREAD_NUM) {
            #pragma unroll 2
            for (int j = 0; j < B; j += THREAD_NUM) {
                int si = tid_y + i;
                int sj = tid_x + j;
                unsigned int tmp = pivot[si][k] + self[k][sj];
                self[si][sj] = min(self[si][sj], tmp);
            }
        }
    }
    
    /* self(k,j) (share -> global) */
    #pragma unroll 2
    for (int k = 0; k < B; k += THREAD_NUM) {
        #pragma unroll 2
        for (int j = 0; j < B; j += THREAD_NUM) {
            int sk = tid_y + k;
            int sj = tid_x + j;

            int global_idx = (pivot_start + sk) * row_size + block_start + sj;
            
            dist[global_idx] = self[sk][sj];
        }
    }
}

__global__ void block_FW_phase3(int row_size, int round, int block_base_x, int block_base_y, unsigned int *dist) {
    __shared__ unsigned int ij[B][B];
    __shared__ unsigned int ik[B][B];
    __shared__ unsigned int kj[B][B];
    
    int block_start_j = (block_base_x + blockIdx.x) * B;
    int block_start_i = (block_base_y + blockIdx.y) * B;
    int block_start_k = round * B;

    int tid_x = threadIdx.x;
    int tid_y = threadIdx.y;

    /* global to share */
    #pragma unroll 2
    for (int x = 0; x < B; x += THREAD_NUM) {
        #pragma unroll 2
        for (int y = 0; y < B; y += THREAD_NUM) {
            int sx = tid_y + x;
            int sy = tid_x + y;

            int g_ij_idx = (block_start_i + sx) * row_size + block_start_j + sy;
            int g_ik_idx = (block_start_i + sx) * row_size + block_start_k + sy;
            int g_kj_idx = (block_start_k + sx) * row_size + block_start_j + sy;

            ij[sx][sy] = dist[g_ij_idx];
            ik[sx][sy] = dist[g_ik_idx];
            kj[sx][sy] = dist[g_kj_idx];
        }
    }

    __syncthreads();
    /* cal */
    #pragma unroll 64
    for (int k = 0; k < B; ++k) {
        #pragma unroll 2
        for (int i = 0; i < B; i += THREAD_NUM) {
            #pragma unroll 2
            for (int j = 0; j < B; j += THREAD_NUM) {
                int si = tid_y + i;
                int sj = tid_x + j;
                unsigned int tmp = ik[si][k] + kj[k][sj];
                ij[si][sj] = min(ij[si][sj], tmp);
            }
        }
    }

    /* i, j (share -> global) */
    #pragma unroll 2
    for (int i = 0; i < B; i += THREAD_NUM) {
        #pragma unroll 2
        for (int j = 0; j < B; j += THREAD_NUM) {
            int si = tid_y + i;
            int sj = tid_x + j;
            
            int global_idx = (block_start_i + si) * row_size + block_start_j + sj;
            
            dist[global_idx] = ij[si][sj];
        }
    }
}

int main(int argc, char* argv[]) {
    unsigned int *G, *d_dist[2];
    omp_lock_t lock[2];

    omp_init_lock(&lock[0]);
    omp_init_lock(&lock[1]);
    size_t pitch;
    G = input(argv[1]);
    printf("v_num: %d, v_orig: %d.\n", v_num, v_orig);
    hipSetDevice(0);
    hipMallocPitch((void **)&d_dist[0], &pitch, v_num * sizeof(unsigned int), v_num);

    hipSetDevice(1);
    hipMallocPitch((void **)&d_dist[1], &pitch, v_num * sizeof(unsigned int), v_num);

    int pitch_row_size = pitch / sizeof(unsigned int);
    int round = ceil(v_num, B);
    dim3 num_threads(THREAD_NUM, THREAD_NUM); // because num of threads per block is 1024
    dim3 p2_num_blocks(round, 1);
    dim3 p3_num_blocks(round, 1);

    #pragma omp parallel num_threads(2) default(shared)
    {
        unsigned int gpu_id = omp_get_thread_num();
        int another_gpu_id = (gpu_id + 1) % 2;
        hipSetDevice(gpu_id);
        hipDeviceEnablePeerAccess(another_gpu_id, 0);

        hipMemcpy2D(d_dist[gpu_id], pitch, G, v_num * sizeof(unsigned int),
                     v_num * sizeof(unsigned int), v_num, hipMemcpyHostToDevice);

        hipStream_t streams[STREAM_SIZE];
        for (int i = 0; i < STREAM_SIZE; ++i) {
            hipStreamCreate(&streams[i]);
        }

        for (int r = 0; r < round - 1; ++r) {
            /* Phase 1 */
            omp_set_lock(&lock[another_gpu_id]);
            printf("(%d)gpu: %d.\n", r, gpu_id);

            block_FW_phase1 <<<1, num_threads>>> (pitch_row_size, r, d_dist[gpu_id]);

            /* Phase 2 */
            block_FW_phase2_row <<<p2_num_blocks, num_threads>>> (pitch_row_size, r, d_dist[gpu_id]);

            block_FW_phase2_column <<<p2_num_blocks, num_threads>>> (pitch_row_size, r, d_dist[gpu_id]);

            /* Phase 3 */
            if (gpu_id == 0) {
                for (int i = 0; i < round / 2; ++i) {
                    if (unlikely(i == r)) {
                        continue;
                    }
                    block_FW_phase3 <<<p3_num_blocks, num_threads>>>
                                    (pitch_row_size, r, 0, i, d_dist[gpu_id]);
                    if (i == r + 1) {
                        // cudaMemcpyPeerAsync(d_dist[another_gpu_id] + i * B * pitch_row_size, another_gpu_id,
                        //                 d_dist[gpu_id] + i * B * pitch_row_size, gpu_id,
                        //                 B * pitch, streams[i % STREAM_SIZE]);
                        hipMemcpyPeer(d_dist[another_gpu_id] + i * B * pitch_row_size, another_gpu_id,
                                        d_dist[gpu_id] + i * B * pitch_row_size, gpu_id,
                                        B * pitch);
                    }
                    
                }
            } else if (gpu_id == 1) {
                for (int i = round / 2; i < round; ++i) {
                    if (unlikely(i == r)) {
                        continue;
                    }
                    block_FW_phase3 <<<p3_num_blocks, num_threads>>>
                                    (pitch_row_size, r, 0, i, d_dist[gpu_id]);
                    
                    if (i == r + 1) {
                        // cudaMemcpyPeerAsync(d_dist[another_gpu_id] + i * B * pitch_row_size, another_gpu_id,
                        //                     d_dist[gpu_id] + i * B * pitch_row_size, gpu_id,
                        //                     B * pitch, streams[i % STREAM_SIZE]);
                        hipMemcpyPeer(d_dist[another_gpu_id] + i * B * pitch_row_size, another_gpu_id,
                                        d_dist[gpu_id] + i * B * pitch_row_size, gpu_id,
                                        B * pitch);
                    }
                }
            }
            hipDeviceSynchronize();
            omp_unset_lock(&lock[gpu_id]);
        }

        /* last round */
        int r = round - 1;
        omp_set_lock(&lock[another_gpu_id]);
        block_FW_phase1 <<<1, num_threads>>> (pitch_row_size, r, d_dist[gpu_id]);

        block_FW_phase2_row <<<p2_num_blocks, num_threads>>> (pitch_row_size, r, d_dist[gpu_id]);

        block_FW_phase2_column <<<p2_num_blocks, num_threads>>> (pitch_row_size, r, d_dist[gpu_id]);

        hipDeviceSynchronize();

        if (gpu_id == 0) {
            for (int i = 0; i < r / 2; ++i) {
                // block_FW_phase3 <<<p3_num_blocks, num_threads, 0, streams[i % STREAM_SIZE]>>>
                //                 (pitch_row_size, r, 0, i, d_dist[gpu_id]);
                
                // cudaMemcpy2DAsync(G + i * B * v_num, v_num * sizeof(unsigned int),
                //                   d_dist[gpu_id] + i * B * pitch_row_size, pitch,
                //                   v_num * sizeof(unsigned int), B, cudaMemcpyDeviceToHost,
                //                   streams[i % STREAM_SIZE]);
                block_FW_phase3 <<<p3_num_blocks, num_threads>>>
                                (pitch_row_size, r, 0, i, d_dist[gpu_id]);
                
                hipMemcpy2D(G + i * B * v_num, v_num * sizeof(unsigned int),
                                  d_dist[gpu_id] + i * B * pitch_row_size, pitch,
                                  v_num * sizeof(unsigned int), B, hipMemcpyDeviceToHost);
            }
        } else if (gpu_id == 1) {
            hipMemcpy2D(G + r * B * v_num, v_num * sizeof(unsigned int),
                         d_dist[gpu_id] + r * B * pitch_row_size, pitch,
                         v_num * sizeof(unsigned int), B, hipMemcpyDeviceToHost);
            for (int i = r / 2; i < r; ++i) {
                // block_FW_phase3 <<<p3_num_blocks, num_threads, 0, streams[i % STREAM_SIZE]>>>
                //                 (pitch_row_size, r, 0, i, d_dist[gpu_id]);
                
                // cudaMemcpy2DAsync(G + i * B * v_num, v_num * sizeof(unsigned int),
                //                   d_dist[gpu_id] + i * B * pitch_row_size, pitch,
                //                   v_num * sizeof(unsigned int), B, cudaMemcpyDeviceToHost,
                //                   streams[i % STREAM_SIZE]);
                block_FW_phase3 <<<p3_num_blocks, num_threads>>>
                                (pitch_row_size, r, 0, i, d_dist[gpu_id]);
                
                hipMemcpy2D(G + i * B * v_num, v_num * sizeof(unsigned int),
                                  d_dist[gpu_id] + i * B * pitch_row_size, pitch,
                                  v_num * sizeof(unsigned int), B, hipMemcpyDeviceToHost);
            }
        }
        omp_unset_lock(&lock[gpu_id]);

        for (int i = 0; i < STREAM_SIZE; ++i) {
            hipStreamDestroy(streams[i]);
        }

    }
    hipDeviceSynchronize();

    omp_destroy_lock(&lock[0]);
    omp_destroy_lock(&lock[1]);
    output(argv[2], G);
    hipHostFree(G);
    hipFree(d_dist[0]);
    hipFree(d_dist[1]);

    return 0;
}