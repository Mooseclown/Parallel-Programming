
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <algorithm>

#define B 64
#define thread_num 32

#define unlikely(x)  __builtin_expect(!!(x), 0)
#define ROUND_UP(x, align) (((int) (x) + (align - 1)) & ~(align - 1))

const int INF = ((1 << 30) - 1);

int v_num, e_num, v_orig;

inline int ceil(int a, int b) { return (a + b - 1) / b; }

static unsigned int *input(const char *filename) {
    FILE *f;
    f = fopen(filename, "rb");
    assert(f);

    fread(&v_orig, sizeof(int), 1, f);
    fread(&e_num, sizeof(int), 1, f);
    v_num = ROUND_UP(v_orig, B);
    unsigned int *G;
    hipHostMalloc(&G, v_num * v_num * sizeof(unsigned int), hipHostMallocDefault);

    /* init shortest-path distance array*/
    std::fill_n(G, v_num * v_num, INF);
    for (int i = 0; i < v_num; ++i) {
        G[i * v_num + i] = 0;
    }

    int *pairs = (int *)malloc(3 * e_num * sizeof(int));
    fread(pairs, sizeof(int), 3 * e_num, f);
    for (int i = 0; i < e_num; ++i) {
        int index = pairs[3 * i] * v_num + pairs[3 * i + 1];
        G[index] = pairs[3 * i + 2];
    }
    free(pairs);
    
    fclose(f);
    return G;
}

void output(char* outFileName, unsigned int *G) {
    FILE *f;
    f = fopen(outFileName, "wb+");
    assert(f);

    for (int i = 0; i < v_orig; ++i) {
        fwrite(G + i * v_num, sizeof(int), v_orig, f);
    }
    fclose(f);
}

__global__ void block_FW_phase1(int row_size, int round, unsigned int *dist) {
    __shared__ unsigned int s_dist[B][B];

    int block_start = round * B;

    int tid_x = threadIdx.x;
    int tid_y = threadIdx.y;

    /* pivot (global -> share) */
    #pragma unroll 2
    for (int i = 0; i < B; i += thread_num) {
        #pragma unroll 2
        for (int j = 0; j < B; j += thread_num) {
            int si = tid_y + i;
            int sj = tid_x + j;

            int global_idx = (block_start + si) * row_size + block_start + sj;

            s_dist[si][sj] = dist[global_idx];
        }
    }

    /* cal */
    #pragma unroll 64
    for (int k = 0; k < B; ++k) {
        __syncthreads();
        #pragma unroll 2
        for (int i = 0; i < B; i += thread_num) {
            #pragma unroll 2
            for (int j = 0; j < B; j += thread_num) {
                int si = tid_y + i;
                int sj = tid_x + j;
                unsigned int tmp = s_dist[si][k] + s_dist[k][sj];
                s_dist[si][sj] = min(s_dist[si][sj], tmp);
            }
        }
    }

    /* pivot (share -> global) */
    #pragma unroll 2
    for (int i = 0; i < B; i += thread_num) {
        #pragma unroll 2
        for (int j = 0; j < B; j += thread_num) {
            int si = tid_y + i;
            int sj = tid_x + j;

            int global_idx = (block_start + si) * row_size + block_start + sj;

            dist[global_idx] = s_dist[si][sj];
        }
    }
}

__global__ void block_FW_phase2_row(int row_size, int round, unsigned int *dist) {    
    __shared__ unsigned int self[B][B];
    __shared__ unsigned int pivot[B][B];

    int pivot_start = round * B;
    int block_start = blockIdx.x * B;

    int tid_x = threadIdx.x;
    int tid_y = threadIdx.y;

    /* global -> share */
    #pragma unroll 2
    for (int x = 0; x < B; x += thread_num) {
        #pragma unroll 2
        for (int y = 0; y < B; y += thread_num) {
            int sx = tid_y + x;
            int sy = tid_x + y;

            int self_idx =  (block_start + sx) * row_size + pivot_start + sy;
            int pivot_idx = (pivot_start + sx) * row_size + pivot_start + sy;

            self[sx][sy] = dist[self_idx];
            pivot[sx][sy] = dist[pivot_idx];
        }
    }

    /* cal */
    #pragma unroll 64
    for (int k = 0; k < B; ++k) {
        __syncthreads();
        #pragma unroll 2
        for (int i = 0; i < B; i += thread_num) {
            #pragma unroll 2
            for (int j = 0; j < B; j += thread_num) {
                int si = tid_y + i;
                int sj = tid_x + j;
                unsigned int tmp = self[si][k] + pivot[k][sj];
                self[si][sj] = min(self[si][sj], tmp);
            }
        }
    }
    
    /* self(i,k) (share -> global) */
    #pragma unroll 2
    for (int i = 0; i < B; i += thread_num) {
        #pragma unroll 2
        for (int k = 0; k < B; k += thread_num) {
            int si = tid_y + i;
            int sk = tid_x + k;

            int global_idx = (block_start + si) * row_size + pivot_start + sk;

            dist[global_idx] = self[si][sk];
        }
    }
}

__global__ void block_FW_phase2_column(int row_size, int round, unsigned int *dist) {
    __shared__ unsigned int self[B][B];
    __shared__ unsigned int pivot[B][B];

    int pivot_start = round * B;
    int block_start = blockIdx.x * B;

    int tid_x = threadIdx.x;
    int tid_y = threadIdx.y;

    /* global -> share */
    #pragma unroll 2
    for (int x = 0; x < B; x += thread_num) {
        #pragma unroll 2
        for (int y = 0; y < B; y += thread_num) {
            int sx = tid_y + x;
            int sy = tid_x + y;

            int self_idx = (pivot_start + sx) * row_size + block_start + sy;
            int pivot_idx = (pivot_start + sx) * row_size + pivot_start + sy;

            self[sx][sy] = dist[self_idx];
            pivot[sx][sy] = dist[pivot_idx];
        }
    }

    /* cal */
    #pragma unroll 64
    for (int k = 0; k < B; ++k) {
        __syncthreads();
        #pragma unroll 2
        for (int i = 0; i < B; i += thread_num) {
            #pragma unroll 2
            for (int j = 0; j < B; j += thread_num) {
                int si = tid_y + i;
                int sj = tid_x + j;
                unsigned int tmp = pivot[si][k] + self[k][sj];
                self[si][sj] = min(self[si][sj], tmp);
            }
        }
    }
    
    /* self(k,j) (share -> global) */
    #pragma unroll 2
    for (int k = 0; k < B; k += thread_num) {
        #pragma unroll 2
        for (int j = 0; j < B; j += thread_num) {
            int sk = tid_y + k;
            int sj = tid_x + j;

            int global_idx = (pivot_start + sk) * row_size + block_start + sj;
            
            dist[global_idx] = self[sk][sj];
        }
    }
}

__global__ void block_FW_phase3(int row_size, int round, int block_base_x, int block_base_y, unsigned int *dist) {
    __shared__ unsigned int ij[B][B];
    __shared__ unsigned int ik[B][B];
    __shared__ unsigned int kj[B][B];
    
    int block_start_j = (block_base_x + blockIdx.x) * B;
    int block_start_i = (block_base_y + blockIdx.y) * B;
    int block_start_k = round * B;

    int tid_x = threadIdx.x;
    int tid_y = threadIdx.y;

    /* global to share */
    #pragma unroll 2
    for (int x = 0; x < B; x += thread_num) {
        #pragma unroll 2
        for (int y = 0; y < B; y += thread_num) {
            int sx = tid_y + x;
            int sy = tid_x + y;

            int g_ij_idx = (block_start_i + sx) * row_size + block_start_j + sy;
            int g_ik_idx = (block_start_i + sx) * row_size + block_start_k + sy;
            int g_kj_idx = (block_start_k + sx) * row_size + block_start_j + sy;

            ij[sx][sy] = dist[g_ij_idx];
            ik[sx][sy] = dist[g_ik_idx];
            kj[sx][sy] = dist[g_kj_idx];
        }
    }

    __syncthreads();
    /* cal */
    #pragma unroll 64
    for (int k = 0; k < B; ++k) {
        #pragma unroll 2
        for (int i = 0; i < B; i += thread_num) {
            #pragma unroll 2
            for (int j = 0; j < B; j += thread_num) {
                int si = tid_y + i;
                int sj = tid_x + j;
                unsigned int tmp = ik[si][k] + kj[k][sj];
                ij[si][sj] = min(ij[si][sj], tmp);
            }
        }
    }

    /* i, j (share -> global) */
    #pragma unroll 2
    for (int i = 0; i < B; i += thread_num) {
        #pragma unroll 2
        for (int j = 0; j < B; j += thread_num) {
            int si = tid_y + i;
            int sj = tid_x + j;
            
            int global_idx = (block_start_i + si) * row_size + block_start_j + sj;
            
            dist[global_idx] = ij[si][sj];
        }
    }
}

int main(int argc, char* argv[]) {
    unsigned int *G, *d_dist;
    size_t pitch;
    G = input(argv[1]);

    hipMallocPitch((void **)&d_dist, &pitch, v_num * sizeof(unsigned int), v_num);

    hipMemcpy2D(d_dist, pitch, G, v_num * sizeof(unsigned int), v_num * sizeof(unsigned int), v_num, hipMemcpyHostToDevice);

    int pitch_row_size = pitch / sizeof(unsigned int);
    int round = ceil(v_num, B);
    dim3 num_threads(thread_num, thread_num); // because num of threads per block is 1024
    dim3 p2_num_blocks(round, 1);
    dim3 p3_num_blocks;
    for (int r = 0; r < round - 1; ++r) {
        int tmp = round - (r + 1);
        
        block_FW_phase1 <<<1, num_threads>>> (pitch_row_size, r, d_dist);

        block_FW_phase2_row <<<p2_num_blocks, num_threads>>> (pitch_row_size, r, d_dist);

        block_FW_phase2_column <<<p2_num_blocks, num_threads>>> (pitch_row_size, r, d_dist);

        // /* Phase 3 */
        // if (r > 0) {
        //     /* 左上 */
        //     p3_num_blocks.x = r;
        //     p3_num_blocks.y = 1;
        //     for (int i = 0; i < r; ++i) {
        //         block_FW_phase3 <<<p3_num_blocks, num_threads>>> (pitch_row_size, r, 0, i, d_dist);
        //     }
        // }
        // if (tmp > 0) {
        //     /* 右下 */
        //     p3_num_blocks.x = tmp;
        //     p3_num_blocks.y = 1;
        //     for (int i = 0; i < tmp; ++i) {
        //         block_FW_phase3 <<<p3_num_blocks, num_threads>>> (pitch_row_size, r, r+1, r+1+i, d_dist);
        //     }
        // }
        // if (r > 0 && tmp > 0) {
        //     /* 左下 */
        //     p3_num_blocks.x = r;
        //     p3_num_blocks.y = 1;
        //     for (int i = 0; i < tmp; ++i) {
        //         block_FW_phase3 <<<p3_num_blocks, num_threads>>> (pitch_row_size, r, 0, r+1+i, d_dist);
        //     }
        //     /* 右上 */
        //     p3_num_blocks.x = tmp;
        //     p3_num_blocks.y = 1;
        //     for (int i = 0; i < r; ++i) {
        //         block_FW_phase3 <<<p3_num_blocks, num_threads>>> (pitch_row_size, r, r+1, i, d_dist);
        //     }
        // }

        /* Phase 3 */
        if (r > 0) {
            /* 左上 */
            p3_num_blocks.x = r;
            p3_num_blocks.y = r;
            block_FW_phase3 <<<p3_num_blocks, num_threads>>> (pitch_row_size, r, 0, 0, d_dist);
        }
        if (tmp > 0) {
            /* 右下 */
            p3_num_blocks.x = tmp;
            p3_num_blocks.y = tmp;
            block_FW_phase3 <<<p3_num_blocks, num_threads>>> (pitch_row_size, r, r+1, r+1, d_dist);
        }
        if (r > 0 && tmp > 0) {
            /* 左下 */
            p3_num_blocks.x = r;
            p3_num_blocks.y = tmp;
            block_FW_phase3 <<<p3_num_blocks, num_threads>>> (pitch_row_size, r, 0, r+1, d_dist);
            /* 右上 */
            p3_num_blocks.x = tmp;
            p3_num_blocks.y = r;
            block_FW_phase3 <<<p3_num_blocks, num_threads>>> (pitch_row_size, r, r+1, 0, d_dist);
        }

    }

    /* last round */
    block_FW_phase1 <<<1, num_threads>>> (pitch_row_size, round -1, d_dist);

    block_FW_phase2_row <<<p2_num_blocks, num_threads>>> (pitch_row_size, round -1, d_dist);

    block_FW_phase2_column <<<p2_num_blocks, num_threads>>> (pitch_row_size, round -1, d_dist);

    /* Phase 3 */
    /* 左上 */
    hipStream_t streams[20];
    for (int i = 0; i < 20; ++i) {
        hipStreamCreate(&streams[i]);
    }

    p3_num_blocks.x = round - 1;
    p3_num_blocks.y = 1;
    for (int i = 0; i < round - 1; ++i) {
        block_FW_phase3 <<<p3_num_blocks, num_threads, 0, streams[i%20]>>> (pitch_row_size, round -1, 0, i, d_dist);
        
        hipMemcpy2DAsync(G + i * B * v_num, v_num * sizeof(unsigned int),
                     d_dist + i * B * pitch_row_size, pitch,
                     v_num * sizeof(unsigned int), B, hipMemcpyDeviceToHost, streams[i%20]);
    }

    for (int i = 0; i < 20; ++i) {
        hipStreamDestroy(streams[i]);
    }

    hipMemcpy2D(G + (round - 1) * B * v_num, v_num * sizeof(unsigned int),
                 d_dist + (round - 1) * B * pitch_row_size, pitch,
                 v_num * sizeof(unsigned int), B, hipMemcpyDeviceToHost);

    //cudaMemcpy2D(G, v_num * sizeof(unsigned int), d_dist, pitch, v_num * sizeof(unsigned int), v_num, cudaMemcpyDeviceToHost);

    hipFree(d_dist);

    output(argv[2], G);
    hipHostFree(G);

    return 0;
}